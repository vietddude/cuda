#include "hip/hip_runtime.h"
#include "./kernel_interface.h"
#include <iostream>
#include <cmath>

__global__ void forward_gpu_tiled(float *output, const float *input, const float *kernel, const float *bias,
                                  const int num_samples, const int output_channel, const int input_channel,
                                  const int height, const int width, const int kernel_size)
{
    int tileSize = height + kernel_size - 1;
    int height_out = height - kernel_size + 1;
    int width_out = width - kernel_size + 1;

    int height_grid = ceil(1.0 * height_out / height);
    int width_grid = ceil(1.0 * width_out / width);

    extern __shared__ float smem[];

    float *x_s = (float *)&smem[0];
    float *k_s = (float *)&smem[tileSize * tileSize];

    int n = blockIdx.x;     // Batch number index
    int m = blockIdx.y;     // Output feature index

    // vertical base out data index for the block
    // blockIdx.z -> number of TILES needed for calculating entire output feature map
    int h_in = (blockIdx.z / width_grid); // TILE's index in output feature map
    // horizontal base out data index for the block
    int w_in = (blockIdx.z % width_grid); // TILE's index in output feature map
    // h0 and w0 used as shorthand for threadIdx.x and threadIdx.y
    int h0 = threadIdx.y;               // index in TILE
    int w0 = threadIdx.x;               // index in TILE
    int h_out = h_in * height + h0;     // real index in output feature map
    int w_out = w_in * width + w0;      // real index in output feature map
    // h_out and w_out is not center point, it's upper left corner point of Input image

    float acc = 0.0f;

    for (int channel = 0; channel < input_channel; channel++)
    {
        if (h0 < kernel_size && w0 < kernel_size)
        {
            k_s[h0 * kernel_size + w0] = kernel[m * (input_channel * kernel_size * kernel_size) +
                                                channel * (kernel_size * kernel_size) +
                                                h0 * kernel_size + w0];
        }
        __syncthreads();
        for (int i = h_out; i < h_in + tileSize; i += height)
        {
            for (int j = w_out; j < w_in + tileSize; j += width)
            {
                if (i - h_in < tileSize && j - w_in < tileSize)
                {
                    x_s[(i - h_in) * tileSize + j - w_in] = input[(n * (input_channel * height * width)) +
                                                                  channel * (height * width) +
                                                                  i * width + j];
                }
            }
        }
        __syncthreads();
        for (int i = 0; i < kernel_size; i++)
        {
            for (int j = 0; j < kernel_size; j++)
            {
                if (h_out < height_out && w_out < width_out)
                {
                    acc += x_s[(h0 + i) * tileSize + w0 + j] * k_s[i * kernel_size + j];
                }
            }
        }
        __syncthreads();
    }
    if (h_out < height_out && w_out < width_out)
    {
        output[n * (output_channel * height_out * width_out) +
               m * (height_out * width_out) +
               h_out * width_out + w_out] = acc + bias[m];
    }
}

__host__ void KernelInterface::forward_kernel(float *output_data, const float *input_data, const float *weight_data, const float *bias_data,
                                              const int num_samples, const int output_channel, const int input_channel,
                                              const int height_in, const int width_in, const int kernel_height)
{
    std::cout << "GPU Convolution layer optimized ver 3." << std::endl;
    const int height_out = height_in - kernel_height + 1;
    const int width_out = width_in - kernel_height + 1;

    // Allocate device memory
    float *device_input, *device_output, *device_weight, *device_bias;
    CHECK(hipMalloc(&device_input, num_samples * input_channel * height_in * width_in * sizeof(float)));              // input features map is input_channel
    CHECK(hipMalloc(&device_output, num_samples * output_channel * height_out * width_out * sizeof(float)));          // output feature map is output_channel
    CHECK(hipMalloc(&device_weight, output_channel * input_channel * kernel_height * kernel_height * sizeof(float))); // input_channel * output_channel filter Maps of size kernel_height * kernel_height
    CHECK(hipMalloc((void **)&device_bias, output_channel * sizeof(float)));
    // Copy input and mask data to device
    CHECK(hipMemcpy(device_input, input_data, num_samples * input_channel * height_in * width_in * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_weight, weight_data, output_channel * input_channel * kernel_height * kernel_height * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(device_bias, bias_data, output_channel * sizeof(float), hipMemcpyHostToDevice));
    //
    dim3 blockSize(height_in, width_in, 1);
    int height_grid = (height_out - 1) / height_in + 1;
    int width_grid = (width_out - 1) / width_in + 1;
    int z = height_grid * width_grid;
    dim3 gridSize(num_samples, output_channel, z);

    size_t smem = ((height_in + kernel_height - 1) * (width_in + kernel_height - 1) + kernel_height * kernel_height) * sizeof(float);
    forward_gpu_tiled<<<gridSize, blockSize, smem >>>(device_output, device_input, device_weight, device_bias, num_samples, output_channel, input_channel, height_in, width_in, kernel_height);
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    // Copy the output back to host
    hipMemcpy(output_data, device_output, num_samples * output_channel * height_out * width_out * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_weight);
}